#include "hip/hip_runtime.h"


#include <iostream>

#include "../helpers/LCommon.h"

using namespace std;


#define N 10000000

#define THREADS_PER_BLOCK 128



__global__ void kernelVectorAdd( float* d_v1, float* d_v2, float* d_v3 )
{
	// note: this is actually less than blockDim.x * gridDim.x
	int tIndx = threadIdx.x + blockIdx.x * blockDim.x;

	while ( tIndx < N )
	{
		d_v3[tIndx] = d_v1[tIndx] + d_v2[tIndx];
		tIndx += blockDim.x * gridDim.x;
	}
}



int main()
{

	float* h_v1 = new float[N];
	float* h_v2 = new float[N];
	float* h_v3 = new float[N];

	for ( int q = 0; q < N; q++ )
	{
		h_v1[q] = ( (float) 2 * q + 1 );
		h_v2[q] = ( (float) 2 * q + 2 );
	}

	float* d_v1;
	float* d_v2;
	float* d_v3;

	hipMalloc( ( void** ) &d_v1, sizeof( float ) * N );
	hipMalloc( ( void** ) &d_v2, sizeof( float ) * N );
	hipMalloc( ( void** ) &d_v3, sizeof( float ) * N );

	hipMemcpy( d_v1, h_v1, sizeof( float ) * N, hipMemcpyHostToDevice );
	hipMemcpy( d_v2, h_v2, sizeof( float ) * N, hipMemcpyHostToDevice );


	kernelVectorAdd<<< 128, 128 >>>( d_v1, d_v2, d_v3 );

	hipMemcpy( h_v3 , d_v3, sizeof( float ) * N, hipMemcpyDeviceToHost );

	free( h_v1 );
	free( h_v2 );
	free( h_v3 );
	hipFree( d_v1 );
	hipFree( d_v2 );
	hipFree( d_v3 );

	return 0;
}